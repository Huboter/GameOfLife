#include "hip/hip_runtime.h"
#include "kernel.h"

#include "hip/hip_runtime.h"
#include ""

__device__ uint8_t calculateNumberOfLivingNeighbours(uint8_t* board, const unsigned int position, const unsigned int numberOfRows, const unsigned int numberOfColumns) {
	uint8_t numberOfLivingNeighbours = 0;

	unsigned int row = position / numberOfColumns;
	unsigned int column = position - (row * numberOfColumns);

	unsigned int topRow = row - 1;
	unsigned int bottomRow = row + 1;
	unsigned int leftColumn = column - 1;
	unsigned int rightColumn = column + 1;

	if (row == 0) {
		topRow = numberOfRows - 1;
	}
	else if (row == numberOfRows - 1) {
		bottomRow = 0;
	}

	if (column == 0) {
		leftColumn = numberOfColumns - 1;
	}
	else if (column == numberOfColumns - 1) {
		rightColumn = 0;
	}

	unsigned int rowAdder = topRow * numberOfColumns;
	numberOfLivingNeighbours += board[leftColumn + rowAdder];
	numberOfLivingNeighbours += board[column + rowAdder];
	numberOfLivingNeighbours += board[rightColumn + rowAdder];

	rowAdder = row * numberOfColumns;
	numberOfLivingNeighbours += board[leftColumn + rowAdder];
	numberOfLivingNeighbours += board[rightColumn + rowAdder];

	rowAdder = bottomRow * numberOfColumns;
	numberOfLivingNeighbours += board[leftColumn + rowAdder];
	numberOfLivingNeighbours += board[column + rowAdder];
	numberOfLivingNeighbours += board[rightColumn + rowAdder];

	return numberOfLivingNeighbours;
}

__global__ void doGenerationKernel(uint8_t* board, uint8_t* tempBoard, const unsigned int numberOfRows, const unsigned int numberOfColumns, const unsigned int boardSize) {
	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int stride = blockDim.x * gridDim.x;

	for (unsigned int i = index; i < boardSize; i += stride) {
		const uint8_t numberOfLivingNeighbours = calculateNumberOfLivingNeighbours(board, i, numberOfRows, numberOfColumns);

		if (board[i] == 0 && numberOfLivingNeighbours == 3) {
			tempBoard[i] = 1;
		}
		else if (numberOfLivingNeighbours != 2 && numberOfLivingNeighbours != 3) {
			tempBoard[i] = 0;
		}
	}
}

void gpgpu::doGeneration(std::vector<uint8_t>& board, const unsigned int numberOfRows, const unsigned int numberOfColumns, const unsigned int numberOfGenerations) {
	const size_t boardSize = numberOfRows * numberOfColumns;
	const size_t boardSizeInByte = boardSize * sizeof(uint8_t);

	uint8_t* cudaBoard;
	uint8_t* cudaTempBoard;

	hipMallocManaged(&cudaBoard, boardSizeInByte);
	hipMallocManaged(&cudaTempBoard, boardSizeInByte);

	for (size_t i = 0; i < boardSize; ++i) {
		cudaBoard[i] = board[i];
		cudaTempBoard[i] = board[i];
	}

	const unsigned int blockSize = 256;
	const unsigned int numberOfBlocks = (boardSize + blockSize - 1) / blockSize;

	for (size_t i = 0; i < numberOfGenerations; ++i) {
		doGenerationKernel<<<numberOfBlocks, blockSize>>>(cudaBoard, cudaTempBoard, numberOfRows, numberOfColumns, boardSize);
		hipDeviceSynchronize();
		hipMemcpy(cudaBoard, cudaTempBoard, boardSizeInByte, hipMemcpyDeviceToDevice);
	}

	for (size_t i = 0; i < boardSize; ++i) {
		board[i] = cudaBoard[i];
	}

	hipFree(cudaTempBoard);
	hipFree(cudaBoard);
}